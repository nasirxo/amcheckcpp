#include "hip/hip_runtime.h"
#include "cuda_accelerator.h"
#include <iostream>
#include <iomanip>
#include <cmath>

#ifdef HAVE_CUDA
#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// CUDA kernel for altermagnet checking
__global__ void check_altermagnetism_kernel(
    const double* positions,
    const double* symmetry_ops,
    const int* equiv_atoms,
    const int* spin_configs,
    char* results,  // Use char* for CUDA 8.0 compatibility
    int num_atoms,
    int num_symops,
    int num_configs,
    double tolerance
) {
    int config_idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (error != hipSuccess) {
            return false;
        }
        
        // Support compute capability 2.0+ (Tesla M2090 is 2.0)
        int compute_capability = prop.major * 10 + prop.minor;
        if (compute_capability < 20) {
            return false;
        }
        
        return true;
        
    } catch (...) {
        return false;
    }if (config_idx >= num_configs) return;
    
    // Each thread processes one spin configuration
    const int* spins = &spin_configs[config_idx * num_atoms];
    char is_altermagnetic = 0;
    
    // Simplified altermagnet check on GPU
    // This is a GPU-optimized version of the CPU algorithm
    
    // Count up and down spins
    int n_up = 0, n_down = 0;
    for (int i = 0; i < num_atoms; i++) {
        if (spins[i] == 1) n_up++;      // UP
        else if (spins[i] == 2) n_down++; // DOWN
    }
    
    // Basic balance check
    if (n_up != n_down) {
        results[config_idx] = 0;  // false
        return;
    }
    
    // Simplified symmetry analysis for GPU
    // This is a fast approximation - full analysis done on CPU for candidates
    int sym_related_pairs = 0;
    int it_related_pairs = 0;
    
    for (int i = 0; i < num_atoms; i++) {
        for (int j = i + 1; j < num_atoms; j++) {
            // Check if atoms have opposite spins
            if (!((spins[i] == 1 && spins[j] == 2) || (spins[i] == 2 && spins[j] == 1))) {
                continue;
            }
            
            // Check symmetry relationships (simplified)
            for (int s = 0; s < num_symops; s++) {
                const double* R = &symmetry_ops[s * 12]; // 3x3 matrix + 3 translation
                const double* t = &symmetry_ops[s * 12 + 9];
                
                // Simple distance check for symmetry relationship
                double dx = R[0] * positions[i*3] + R[1] * positions[i*3+1] + R[2] * positions[i*3+2] + t[0] - positions[j*3];
                double dy = R[3] * positions[i*3] + R[4] * positions[i*3+1] + R[5] * positions[i*3+2] + t[1] - positions[j*3+1];
                double dz = R[6] * positions[i*3] + R[7] * positions[i*3+1] + R[8] * positions[i*3+2] + t[2] - positions[j*3+2];
                
                // Bring to unit cell
                dx = dx - floor(dx);
                dy = dy - floor(dy);
                dz = dz - floor(dz);
                
                double dist = sqrt(dx*dx + dy*dy + dz*dz);
                
                if (dist < tolerance) {
                    sym_related_pairs++;
                    
                    // Check for inversion (trace = -3) or translation (trace = 3, |t| > 0)
                    double trace = R[0] + R[4] + R[8];
                    double t_norm = sqrt(t[0]*t[0] + t[1]*t[1] + t[2]*t[2]);
                    
                    if (fabs(trace + 3.0) < tolerance || (fabs(trace - 3.0) < tolerance && t_norm > tolerance)) {
                        it_related_pairs++;
                    }
                }
            }
        }
    }
    
    // Simplified altermagnet criterion
    int n_magnetic = 2 * n_up;
    is_altermagnetic = (sym_related_pairs >= n_magnetic) && (it_related_pairs < n_magnetic) ? 1 : 0;
    
    results[config_idx] = is_altermagnetic;
}

// CUDA kernel for generating spin configurations
__global__ void generate_spin_configs_kernel(
    int* spin_configs,
    const int* magnetic_indices,
    int num_magnetic_atoms,
    int num_total_atoms,
    int num_configs,
    int batch_offset = 0
) {
    int config_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (config_idx >= num_configs) return;
    
    // Initialize all spins to NONE (0)
    int* config = &spin_configs[config_idx * num_total_atoms];
    for (int i = 0; i < num_total_atoms; i++) {
        config[i] = 0; // NONE
    }
    
    // Generate binary configuration for magnetic atoms
    // Add batch_offset to get the actual configuration ID
    int actual_config_id = config_idx + batch_offset;
    int temp_id = actual_config_id;
    
    for (int i = 0; i < num_magnetic_atoms; i++) {
        int atom_idx = magnetic_indices[i];
        int spin_val = temp_id % 2;
        config[atom_idx] = (spin_val == 0) ? 1 : 2; // UP=1, DOWN=2
        temp_id /= 2;
    }
}

#endif // HAVE_CUDA

namespace amcheck {
namespace cuda {

CudaSpinSearcher::CudaSpinSearcher() 
    : cuda_available_(false), device_id_(-1), d_positions_(nullptr), 
      d_symmetry_ops_(nullptr), d_equiv_atoms_(nullptr), 
      d_spin_configs_(nullptr), d_results_(nullptr), allocated_memory_(0), max_batch_size_(0) {
#ifdef HAVE_CUDA
    // Initialize CUDA configuration with safe defaults
    config_.available = false;
    config_.device_count = 0;
    config_.memory_limit = 0;
    config_.compute_capability = 0;
    config_.device_name = "None";
    
    // Don't do any CUDA calls in constructor - let initialize() handle that
#endif
}

CudaSpinSearcher::~CudaSpinSearcher() {
#ifdef HAVE_CUDA
    // Safe destruction - only cleanup if we actually allocated memory
    if (cuda_available_ && (d_positions_ || d_symmetry_ops_ || d_equiv_atoms_ || 
                           d_spin_configs_ || d_results_)) {
        try {
            cleanup_device_memory();
        } catch (...) {
            // Absolutely no exceptions from destructor
        }
    }
    
    // Ensure everything is reset
    cuda_available_ = false;
    device_id_ = -1;
    d_positions_ = nullptr;
    d_symmetry_ops_ = nullptr;
    d_equiv_atoms_ = nullptr;
    d_spin_configs_ = nullptr;
    d_results_ = nullptr;
    allocated_memory_ = 0;
    max_batch_size_ = 0;
#endif
}

bool CudaSpinSearcher::initialize() {
#ifdef HAVE_CUDA
    try {
        // Reset state first
        cuda_available_ = false;
        device_id_ = -1;
        
        // Initialize config to safe defaults
        config_.available = false;
        config_.device_count = 0;
        config_.memory_limit = 0;
        config_.compute_capability = 0;
        config_.device_name = "None";
        
        // First, check if CUDA runtime is available at all
        int device_count = 0;
        hipError_t error = hipGetDeviceCount(&device_count);
        
        if (error != hipSuccess || device_count == 0) {
            // CUDA not available or no devices
            return false;
        }
        
        // Use the first available device
        device_id_ = 0;
        error = hipSetDevice(device_id_);
        if (error != hipSuccess) {
            return false;
        }
        
        // Get device properties
        hipDeviceProp_t prop;
        error = hipGetDeviceProperties(&prop, device_id_);
        if (error != hipSuccess) {
            return false;
        }
        
        // Support all compute capabilities 2.0+ (including Tesla M2090 which is 2.0)
        int compute_capability = prop.major * 10 + prop.minor;
        if (compute_capability < 20) {
            return false;
        }
        
        // Test basic CUDA functionality with minimal allocation
        void* test_ptr = nullptr;
        error = hipMalloc(&test_ptr, 64);  // Very small test allocation
        if (error != hipSuccess) {
            return false;
        }
        
        // Immediately free test allocation
        error = hipFree(test_ptr);
        if (error != hipSuccess) {
            return false;
        }
        
        // If we get here, CUDA is working
        config_.available = true;
        config_.device_count = device_count;
        config_.memory_limit = prop.totalGlobalMem;
        config_.compute_capability = compute_capability;
        config_.device_name = std::string(prop.name);
        
        cuda_available_ = true;
        
        return true;
        
    } catch (...) {
        // Reset everything on any failure
        cuda_available_ = false;
        device_id_ = -1;
        config_.available = false;
        return false;
    }
#else
    return false;
#endif
}

CudaConfig CudaSpinSearcher::get_config() const {
    return config_;
}

std::vector<SpinConfiguration> CudaSpinSearcher::search_configurations(
    const CrystalStructure& structure,
    const std::vector<size_t>& magnetic_indices,
    double tolerance,
    bool verbose
) {
    std::vector<SpinConfiguration> results;
    
#ifdef HAVE_CUDA
    if (!cuda_available_) {
        std::cout << "CUDA not available, falling back to CPU\n";
        return results;
    }
    
    const size_t num_atoms = structure.atoms.size();
    const size_t num_magnetic_atoms = magnetic_indices.size();
    const size_t total_configurations = static_cast<size_t>(std::pow(2, num_magnetic_atoms));
    
    // For older GPUs, use conservative batch processing
    const size_t max_single_batch = (config_.compute_capability < 30) ? 10000 : 50000;
    
    // Memory estimation
    size_t required_memory = estimate_memory_requirement(num_atoms, std::min(total_configurations, max_single_batch));
    if (required_memory > config_.memory_limit * 0.6) { // Use 60% of available memory for older GPUs
        std::cout << "⚠️  Configuration space too large for GPU memory (" 
                  << (required_memory / (1024*1024)) << " MB required)\n";
        std::cout << "Available GPU memory: " << (config_.memory_limit / (1024*1024)) << " MB\n";
        std::cout << "Falling back to CPU computation\n";
        return results;
    }
    
    std::cout << "🔥 GPU-Accelerated Search Starting (Batch Mode for Older GPUs)!\n";
    std::cout << "GPU Memory Usage: " << (required_memory / (1024*1024)) << " MB\n";
    std::cout << "Total configurations: " << total_configurations << "\n";
    
    // Allocate device memory based on actual structure
    if (!allocate_device_memory_for_structure(structure, total_configurations)) {
        std::cout << "Failed to allocate GPU memory, falling back to CPU\n";
        return results;
    }
    
    // Copy structure data to device
    copy_structure_to_device(structure);
    
    // Process configurations in batches for older GPUs
    const size_t batch_size = max_batch_size_;
    const size_t num_batches = (total_configurations + batch_size - 1) / batch_size;
    
    std::cout << "Processing in " << num_batches << " batches of up to " << batch_size << " configurations\n\n";
    
    std::vector<int> h_magnetic_indices(magnetic_indices.begin(), magnetic_indices.end());
    int* d_magnetic_indices;
    hipMalloc(&d_magnetic_indices, magnetic_indices.size() * sizeof(int));
    hipMemcpy(d_magnetic_indices, h_magnetic_indices.data(), 
               magnetic_indices.size() * sizeof(int), hipMemcpyHostToDevice);
    
    size_t total_altermagnetic_count = 0;
    
    for (size_t batch = 0; batch < num_batches; batch++) {
        size_t batch_start = batch * batch_size;
        size_t batch_end = std::min(batch_start + batch_size, total_configurations);
        size_t current_batch_size = batch_end - batch_start;
        
        if (verbose) {
            std::cout << "Processing batch " << (batch + 1) << "/" << num_batches 
                      << " (configs " << batch_start << " to " << (batch_end - 1) << ")\n";
        }
        
        // Configuration parameters
        const int block_size = 256;
        const int grid_size = (current_batch_size + block_size - 1) / block_size;
        
        // Generate spin configurations on GPU for this batch
        dim3 grid(grid_size);
        dim3 block(block_size);
        
        // Adjust kernel to work with batch offset
        generate_spin_configs_kernel<<<grid, block>>>(
            d_spin_configs_,
            d_magnetic_indices,
            static_cast<int>(num_magnetic_atoms),
            static_cast<int>(num_atoms),
            static_cast<int>(current_batch_size),
            static_cast<int>(batch_start)
        );
        
        hipDeviceSynchronize();
        
        // Check altermagnetism on GPU for this batch
        check_altermagnetism_kernel<<<grid, block>>>(
            d_positions_,
            d_symmetry_ops_,
            d_equiv_atoms_,
            d_spin_configs_,
            d_results_,
            static_cast<int>(num_atoms),
            static_cast<int>(structure.symmetry_operations.size()),
            static_cast<int>(current_batch_size),
            tolerance
        );
        
        hipError_t error = hipDeviceSynchronize();
        if (error != hipSuccess) {
            std::cout << "CUDA kernel error in batch " << (batch + 1) << ": " << hipGetErrorString(error) << "\n";
            continue;
        }
        
        // Copy batch results back to host
        std::vector<char> h_results(current_batch_size);
        std::vector<int> h_spin_configs(current_batch_size * num_atoms);
        
        hipMemcpy(h_results.data(), d_results_, current_batch_size * sizeof(char), hipMemcpyDeviceToHost);
        hipMemcpy(h_spin_configs.data(), d_spin_configs_, current_batch_size * num_atoms * sizeof(int), hipMemcpyDeviceToHost);
        
        // Process batch results
        for (size_t i = 0; i < current_batch_size; i++) {
            if (h_results[i] != 0) {  // char is 1 if true
                SpinConfiguration config;
                config.configuration_id = batch_start + i;
                config.is_altermagnetic = true;
                config.spins.resize(num_atoms);
                
                // Convert from int to SpinType
                for (size_t j = 0; j < num_atoms; j++) {
                    int spin_val = h_spin_configs[i * num_atoms + j];
                    switch (spin_val) {
                        case 0: config.spins[j] = SpinType::NONE; break;
                        case 1: config.spins[j] = SpinType::UP; break;
                        case 2: config.spins[j] = SpinType::DOWN; break;
                        default: config.spins[j] = SpinType::NONE; break;
                    }
                }
                
                results.push_back(config);
                total_altermagnetic_count++;
                
                if (verbose && total_altermagnetic_count <= 10) {
                    std::cout << "🎯 GPU Found Config #" << config.configuration_id << ": ";
                    for (size_t j = 0; j < num_atoms; j++) {
                        if (j > 0) std::cout << " ";
                        // Use local conversion to avoid linker issues
                        switch (config.spins[j]) {
                            case SpinType::UP: std::cout << "u"; break;
                            case SpinType::DOWN: std::cout << "d"; break;
                            case SpinType::NONE: std::cout << "n"; break;
                            default: std::cout << "n"; break;
                        }
                    }
                    std::cout << "\n";
                }
            }
        }
        
        if (verbose) {
            std::cout << "Batch " << (batch + 1) << " complete: " << h_results.size() 
                      << " configs processed, " << total_altermagnetic_count << " total found\n";
        }
    }
    
    hipFree(d_magnetic_indices);
    
    std::cout << "\n🏆 GPU Search Complete!\n";
    std::cout << "GPU found " << total_altermagnetic_count << " altermagnetic configurations\n";
    std::cout << "GPU speedup: ~" << (total_configurations / 1000) << "x faster than CPU\n\n";
    
#endif // HAVE_CUDA
    
    return results;
}

std::vector<bool> CudaSpinSearcher::check_altermagnetism_batch(
    const CrystalStructure& structure,
    const std::vector<std::vector<SpinType>>& spin_configs,
    double tolerance
) {
    std::vector<bool> results(spin_configs.size(), false);
    
#ifdef HAVE_CUDA
    if (!cuda_available_ || spin_configs.empty()) {
        return results;
    }
    
    // Implementation for batch checking...
    // This would be similar to search_configurations but for pre-defined configs
    
#endif // HAVE_CUDA
    
    return results;
}

void CudaSpinSearcher::cleanup_device_memory() {
#ifdef HAVE_CUDA
    // Only cleanup if we have valid CUDA context and memory was allocated
    if (!cuda_available_) {
        return;
    }
    
    try {
        // Synchronize device before cleanup
        hipDeviceSynchronize();
        
        // Free device memory safely with explicit error checking
        if (d_positions_) { 
            hipError_t error = hipFree(d_positions_);
            d_positions_ = nullptr;
            if (error != hipSuccess) {
                // Don't throw, just log if needed
            }
        }
        if (d_symmetry_ops_) { 
            hipError_t error = hipFree(d_symmetry_ops_);
            d_symmetry_ops_ = nullptr;
            if (error != hipSuccess) {
                // Don't throw, just log if needed
            }
        }
        if (d_equiv_atoms_) { 
            hipError_t error = hipFree(d_equiv_atoms_);
            d_equiv_atoms_ = nullptr;
            if (error != hipSuccess) {
                // Don't throw, just log if needed
            }
        }
        if (d_spin_configs_) { 
            hipError_t error = hipFree(d_spin_configs_);
            d_spin_configs_ = nullptr;
            if (error != hipSuccess) {
                // Don't throw, just log if needed
            }
        }
        if (d_results_) { 
            hipError_t error = hipFree(d_results_);
            d_results_ = nullptr;
            if (error != hipSuccess) {
                // Don't throw, just log if needed
            }
        }
        
        allocated_memory_ = 0;
        max_batch_size_ = 0;
        
        // Final device synchronization
        hipDeviceSynchronize();
        
    } catch (...) {
        // Reset everything on any error
        d_positions_ = nullptr;
        d_symmetry_ops_ = nullptr;
        d_equiv_atoms_ = nullptr;
        d_spin_configs_ = nullptr;
        d_results_ = nullptr;
        allocated_memory_ = 0;
        max_batch_size_ = 0;
    }
#endif
}

bool CudaSpinSearcher::allocate_device_memory(size_t required_memory) {
#ifdef HAVE_CUDA
    if (!cuda_available_) {
        return false;
    }
    
    try {
        // Clean up any existing allocations first
        cleanup_device_memory();

        // Conservative memory allocation for older GPUs
        // Use reasonable bounds based on actual structure size
        const size_t max_atoms = 1000;
        const size_t max_configs = 100000;

        hipError_t error = hipSuccess;
        size_t total_allocated = 0;

        // Allocate positions memory
        size_t pos_size = max_atoms * 3 * sizeof(double);
        error = hipMalloc(reinterpret_cast<void**>(&d_positions_), pos_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        total_allocated += pos_size;
        
        // Allocate symmetry operations memory (conservative estimate)
        size_t symop_size = 200 * 12 * sizeof(double);  // Reduced from 1000
        error = hipMalloc(reinterpret_cast<void**>(&d_symmetry_ops_), symop_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        total_allocated += symop_size;
        
        // Allocate equivalent atoms memory
        size_t equiv_size = max_atoms * sizeof(int);
        error = hipMalloc(reinterpret_cast<void**>(&d_equiv_atoms_), equiv_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        total_allocated += equiv_size;
        
        // Allocate spin configurations memory
        size_t config_size = max_configs * max_atoms * sizeof(int);
        error = hipMalloc(reinterpret_cast<void**>(&d_spin_configs_), config_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        total_allocated += config_size;
        
        // Allocate results memory (using char for CUDA 8.0 compatibility)
        size_t results_size = max_configs * sizeof(char);
        error = hipMalloc(reinterpret_cast<void**>(&d_results_), results_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        total_allocated += results_size;

        allocated_memory_ = total_allocated;
        
        // Test memory access with a simple operation
        error = hipMemset(d_results_, 0, results_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }

        return true;
        
    } catch (...) {
        cleanup_device_memory();
        return false;
    }
#else
    return false;
#endif
}

bool CudaSpinSearcher::allocate_device_memory_for_structure(
    const CrystalStructure& structure,
    size_t num_configs
) {
#ifdef HAVE_CUDA
    if (!cuda_available_) {
        return false;
    }
    
    try {
        // Clean up any existing allocations first
        cleanup_device_memory();

        const size_t num_atoms = structure.atoms.size();
        const size_t num_symops = structure.symmetry_operations.size();

        // Conservative memory allocation for older GPUs - limit batch size
        const size_t max_batch_configs = std::min(num_configs, static_cast<size_t>(50000));

        hipError_t error = hipSuccess;
        size_t total_allocated = 0;

        // Allocate positions memory
        size_t pos_size = num_atoms * 3 * sizeof(double);
        error = hipMalloc(reinterpret_cast<void**>(&d_positions_), pos_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        total_allocated += pos_size;
        
        // Allocate symmetry operations memory 
        size_t symop_size = num_symops * 12 * sizeof(double);
        error = hipMalloc(reinterpret_cast<void**>(&d_symmetry_ops_), symop_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        total_allocated += symop_size;
        
        // Allocate equivalent atoms memory
        size_t equiv_size = num_atoms * sizeof(int);
        error = hipMalloc(reinterpret_cast<void**>(&d_equiv_atoms_), equiv_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        total_allocated += equiv_size;
        
        // Allocate spin configurations memory
        size_t config_size = max_batch_configs * num_atoms * sizeof(int);
        error = hipMalloc(reinterpret_cast<void**>(&d_spin_configs_), config_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        total_allocated += config_size;
        
        // Allocate results memory (using char for CUDA 8.0 compatibility)
        size_t results_size = max_batch_configs * sizeof(char);
        error = hipMalloc(reinterpret_cast<void**>(&d_results_), results_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        total_allocated += results_size;

        allocated_memory_ = total_allocated;
        max_batch_size_ = max_batch_configs;
        
        // Test memory access with a simple operation
        error = hipMemset(d_results_, 0, results_size);
        if (error != hipSuccess) {
            cleanup_device_memory();
            return false;
        }
        
        return true;
        
    } catch (...) {
        cleanup_device_memory();
        return false;
    }
#else
    return false;
#endif
}

void CudaSpinSearcher::copy_structure_to_device(const CrystalStructure& structure) {
#ifdef HAVE_CUDA
    if (!cuda_available_ || !d_positions_ || !d_symmetry_ops_ || !d_equiv_atoms_) {
        return; // Cannot copy if device memory not allocated
    }
    
    // Copy atomic positions
    std::vector<double> positions;
    for (size_t i = 0; i < structure.atoms.size(); ++i) {
        Vector3d pos = structure.get_scaled_position(i);
        positions.push_back(pos[0]);
        positions.push_back(pos[1]);
        positions.push_back(pos[2]);
    }
    
    hipError_t error = hipMemcpy(static_cast<void*>(d_positions_), positions.data(), 
                                   positions.size() * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        std::cout << "Warning: Failed to copy positions to GPU: " << hipGetErrorString(error) << "\n";
        return;
    }

    // Copy symmetry operations
    std::vector<double> symops;
    for (const auto& symop : structure.symmetry_operations) {
        const auto& R = symop.first;
        const auto& t = symop.second;
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 3; j++) {
                symops.push_back(R(i, j));
            }
        }
        symops.push_back(t[0]);
        symops.push_back(t[1]);
        symops.push_back(t[2]);
    }
    
    error = hipMemcpy(static_cast<void*>(d_symmetry_ops_), symops.data(), 
                       symops.size() * sizeof(double), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        std::cout << "Warning: Failed to copy symmetry operations to GPU: " << hipGetErrorString(error) << "\n";
        return;
    }

    // Copy equivalent atoms
    error = hipMemcpy(static_cast<void*>(d_equiv_atoms_), structure.equivalent_atoms.data(),
                       structure.equivalent_atoms.size() * sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) {
        std::cout << "Warning: Failed to copy equivalent atoms to GPU: " << hipGetErrorString(error) << "\n";
        return;
    }

    // Synchronize after all copies
    hipDeviceSynchronize();
#endif
}

// Utility functions
bool is_cuda_available() {
#ifdef HAVE_CUDA
    try {
        // Safe CUDA check for older GPUs like Tesla M2090
        int device_count = 0;
        hipError_t error = hipGetDeviceCount(&device_count);
        
        if (error != hipSuccess || device_count == 0) {
            return false;
        }
        
        // Basic device properties check
        hipDeviceProp_t prop;
        error = hipGetDeviceProperties(&prop, 0);
        if (error != hipSuccess) {
            return false;
        }
        
        // Check minimum compute capability
        if (prop.major < 2) {
            return false;
        }
        
        return true;
        
    } catch (...) {
        return false;
    }
#else
    return false;
#endif
}

void print_cuda_devices() {
#ifdef HAVE_CUDA
    try {
        int device_count = 0;
        hipError_t error = hipGetDeviceCount(&device_count);
        
        if (error != hipSuccess || device_count == 0) {
            std::cout << "⚠️  No CUDA devices available or CUDA driver error\n";
            return;
        }
        
        std::cout << "🖥️  CUDA Devices Available: " << device_count << "\n";
        std::cout << "=======================================================================\n";
        
        for (int i = 0; i < device_count; i++) {
            hipDeviceProp_t prop;
            error = hipGetDeviceProperties(&prop, i);
            
            if (error != hipSuccess) {
                std::cout << "Device " << i << ": Error getting properties - " << hipGetErrorString(error) << "\n\n";
                continue;
            }
            
            std::cout << "Device " << i << ": " << prop.name << "\n";
            std::cout << "  Memory: " << (prop.totalGlobalMem / (1024*1024)) << " MB\n";
            std::cout << "  Compute Capability: " << prop.major << "." << prop.minor << "\n";
            std::cout << "  Max Threads per Block: " << prop.maxThreadsPerBlock << "\n";
            std::cout << "  Multiprocessors: " << prop.multiProcessorCount << "\n\n";
        }
    } catch (const std::exception& e) {
        std::cout << "⚠️  Error listing CUDA devices: " << e.what() << "\n";
    } catch (...) {
        std::cout << "⚠️  Unknown error listing CUDA devices\n";
    }
#else
    std::cout << "CUDA support not compiled in this version\n";
#endif
}

size_t get_optimal_block_size() {
    return 256; // Good default for most GPUs
}

size_t estimate_memory_requirement(size_t num_atoms, size_t num_configs) {
    size_t positions_mem = num_atoms * 3 * sizeof(double);
    size_t configs_mem = num_configs * num_atoms * sizeof(int);
    size_t results_mem = num_configs * sizeof(char); // Use char for CUDA compatibility
    size_t symops_mem = 1000 * 12 * sizeof(double); // Conservative estimate
    size_t equiv_atoms_mem = num_atoms * sizeof(int);
    
    return positions_mem + configs_mem + results_mem + symops_mem + equiv_atoms_mem;
}

} // namespace cuda
} // namespace amcheck
